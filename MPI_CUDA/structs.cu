#include "hip/hip_runtime.h"
//
// Created by robgrzel on 04.06.17.
//
#include "structs.cuh"
#include "OElemCylinder.h"

__host__ __device__ void printf_resultsij_and_id( int id, const AResultSystemStep_t& prev, const AResultSystemStep_t& res ) {
	printf( "prev blockId:%d,ddot_z=%lf, dot_z=%lf,z=%lf,theta=%lf,eta=%lf,u_z=%lf,dot_u_z=%lf,F_damp=%lf, F_drag=%lf,F_prevt=%lf, F_wave=%lf,F=%lf\n",
	        id, prev.ddot_z, prev.dot_z, prev.z, prev.theta, prev.eta, prev.u_z, prev.dot_u_z, prev.F_damp, prev.F_drag, prev.F_rest, prev.F_wave, prev.F
	      );
	printf( "curr blockId:%d,ddot_z=%lf, dot_z=%lf,z=%lf,theta=%lf,eta=%lf,u_z=%lf,dot_u_z=%lf,F_damp=%lf, F_drag=%lf,F_rest=%lf, F_wave=%lf,F=%lf\n",
	        id, res.ddot_z, res.dot_z, res.z, res.theta, res.eta, res.u_z, res.dot_u_z, res.F_damp, res.F_drag, res.F_rest, res.F_wave, res.F
	      );
}


__host__ __device__ void printf_result_and_id( int id, const AResultSystemStep_t& res ) {
	printf( "blockId:%d,ddot_z=%lf, dot_z=%lf,z=%lf,theta=%lf,eta=%lf,u_z=%lf,dot_u_z=%lf,F_damp=%lf, F_drag=%lf,F_rest=%lf, F_wave=%lf,F=%lf\n",
	        id, res.ddot_z, res.dot_z, res.z, res.theta, res.eta, res.u_z, res.dot_u_z, res.F_damp, res.F_drag, res.F_rest, res.F_wave, res.F
	      );
}

__host__ __device__ void printf_results( const AResultSystemStep_t& res ) {
	printf( "ddot_z=%lf, dot_z=%lf,z=%lf,theta=%lf,eta=%lf,u_z=%lf,dot_u_z=%lf,F_damp=%lf, F_drag=%lf,F_rest=%lf, F_wave=%lf,F=%lf\n",
	        res.ddot_z, res.dot_z, res.z, res.theta, res.eta, res.u_z, res.dot_u_z, res.F_damp, res.F_drag, res.F_rest,
	        res.F_wave, res.F
	      );
}


__global__ void kernelPerstep(
		AResultSystemStep_t* d_result, const AWavePuls_t* d_wave, const AElem_t elem,
		const ACylinderFloating_t cons, const int i
) {

	const int id = blockDim.x * blockIdx.x + threadIdx.x;

	const AWavePuls_t t_wave = d_wave[id];



	const AResultSystemStep_t t_result = d_result[id];//from global memory,will not change,for the data source of next data

	d_result[id].calc( t_result, t_wave, elem, cons, i );

	if (id<10) {
		printf_resultsij_and_id( id, t_result, d_result[id] );

	}
}

void echoError( hipError_t& err, const char* strs ) {
	char a[255];
	if (err != hipSuccess) {
		strncpy( a, strs, 255 );
		fprintf( stderr, "Failed to %s,errorCode %s", a, hipGetErrorString( err ) );
		exit( EXIT_FAILURE );
	}
}

//this is useful ,all generators and methods are inside

__host__ __device__ AResultSystemStep_t::AResultSystemStep_t( ) : ddot_z( 0 ), dot_z( 0 ), z( 0 ), theta( 0 ), eta( 0 ),
                                                                  u_z( 0 ), dot_u_z( 0 ), F_damp( 0 ), F_drag( 0 ),
                                                                  F_rest( 0 ), F_wave( 0 ), F_added2( 0 ), F( 0 ) {
};

__host__ __device__ AResultSystemStep_t::AResultSystemStep_t(
		const double ddot_z, const double dot_z, const double z, \
                                const double theta, const double eta, const double u_z, \
                                const double dot_u_z, const double F_damp, const double F_drag, \
                                const double F_rest, const double F_wave, double F_added2, double F
) : ddot_z( ddot_z ), dot_z( dot_z ), z( z ), theta( theta ), eta( eta ),
    u_z( u_z ), dot_u_z( dot_u_z ), F_damp( F_damp ), F_drag( F_drag ),
    F_rest( F_rest ), F_wave( F_wave ), F_added2( F_added2 ), F( F ) {
}

__host__ __device__ void AResultSystemStep_t::set_initial( const double ddot_z_, const double dot_z_, const double z_, double F_ ) {
	//values needed at t=1*dt;
	ddot_z   = ddot_z_;
	dot_z    = dot_z_;
	z        = z_;
	F        = F_;
	//values not needed at t=1*dt;
	theta    = 0;
	eta      = 0;
	u_z      = 0;
	dot_u_z  = 0;
	F_damp   = 0;
	F_drag   = 0;
	F_rest   = 0;
	F_wave   = 0;
	F_added2 = 0;

};


__host__ __device__ AResultSystemStep_t AResultSystemStep_t::operator =( const AResultSystemStep_t& a ) {
	return AResultSystemStep_t( a );
}

__host__ __device__ AResultSystemStep_t AResultSystemStep_t::operator +( const AResultSystemStep_t& a ) {//DEVICE
	return AResultSystemStep_t( a ); //TODO PROPER SUM OF WAVES
}

__host__ __device__ AResultSystemStep_t& AResultSystemStep_t::operator +=( const AResultSystemStep_t& a ) {

	//*TODO proper integration over waves
	ddot_z += a.ddot_z;//
	dot_z += a.dot_z;//
	z += a.z; //

	//theta += a.theta;

	eta += a.eta;
	u_z += a.u_z;
	dot_u_z += a.dot_u_z;

	F_damp += a.F_damp;
	F_drag += a.F_drag;
	F_rest += a.F_rest;
	F_wave += a.F_wave;
	F_added2 += a.F_added2;
	F += a.F;//
	// */
	return *this;

}


__host__ __device__ void AResultSystemStep_t::calc(
		const AResultSystemStep_t& prev, const AWavePuls_t& wave, const AElem_t& elem,
		const ACylinderFloating_t& cons, const int i
) {
	ddot_z = (prev.F - prev.F_added2) / cons.coefMass / elem.m;
	dot_z  = prev.dot_z + ddot_z * cons.dt;
	z      = prev.z + 0.5 * (prev.dot_z + dot_z) * cons.dt;

	theta = i * cons.dt * wave.om - wave.eps;
	eta   = wave.amp * cos( theta );

	u_z     = wave.om * wave.amp * exp( wave.k * (z - elem.d) ) * sin( theta );
	dot_u_z = pow( wave.om, 2 ) * wave.amp * exp( wave.k * (z - elem.d) ) * cos( theta );

	F_damp   = -dot_z * cons.damp33;
	F_drag   = 0.5 * cons.Rho * cons.areaProjected * cons.coefDrag * fabs( u_z - dot_z ) * (u_z - dot_z);
	F_rest   = -1 * cons.Rho * cons.Grav * cons.areaProjected * z;
	F_added2 = elem.m * (1 + cons.coefAddmass) * dot_u_z;
	F_wave   = cons.Grav * cons.Rho * cons.areaProjected * eta;
	F        = F_damp + F_drag + F_rest + F_added2 + F_wave;
}

__global__ void kernelReduceWave( AResultSystemStep_t* d_result, AResultSystemStep_t* g_blockResult ) {


	size_t tinb = blockDim.x;//how many threads in one block
	size_t idx  = blockIdx.x * blockDim.x + threadIdx.x;
	size_t tid  = threadIdx.x;


	extern __shared__ AResultSystemStep_t t_Result[];


	t_Result[tid] = d_result[idx];

	__syncthreads();

	for ( unsigned int s = 1; s < tinb; s *= 2 ) {
		if (tid % (2 * s) == 0) {
			t_Result[tid] += t_Result[tid + s];
		}
		__syncthreads();
	}
	if (tid == 0) {
		g_blockResult[blockIdx.x] = t_Result[0];//this works, so it means the problem is overwrite operator+= doesn't
	}
}

/*
AElem         oed;
AWavePuls_t       owd;
OElemCylinder oec;
OElem         oed;

*/

OElemCylinder oec;
AWavePuls_t       awd;
OWave         ow;
OTime         ot;


AResultSystem_t::~AResultSystem_t( ) {
	printf( "delete AResultSystem_t::AResultSystem_t (0, rank=%d, %d, %d, %d)\n", myrank, numSteps, numElems, numWaves );

	if (myrank) {
	//	free1d(&resultsInitial);
	//	free1d(&resultsBlocks);
		resultsInitial = NULL;
		resultsBlocks  = NULL;

	} else {
	}

	//free2d( &results );
	results = NULL;

	printf( "delete AResultSystem_t::AResultSystem_t (0, rank=%d, %d, %d, %d)\n", myrank, numSteps, numElems, numWaves );

}

template <typename T>
__host__ __device__ void print_arr( T* a, int N ) {
	printf( "A[%d/%d]=%f\n", 0, N, a[0] );
	for ( int i = 1; i < N; i++ ) {
		printf( "A[%d/%d]=%f\n", i, N, a[i] );

	}
}

AResultSystem_t::AResultSystem_t( int myrank, int numElems, int numSteps, int numWaves, double t0, double t1 ) :
		myrank( myrank ), numElems( numElems ), numSteps( numSteps ), numWaves( numWaves ),
		size( numElems * numSteps ), t0( t0 ), t1( t1 ), dt( (t1 - t0) / numSteps ) {
	printf( "AResultSystem_t::AResultSystem_t (rank=%d, %d, %d, %d)\n", myrank, numSteps, numElems, numWaves );

	malloc2d( &results, numElems, numSteps );


	if (myrank) {
		printf( "AResultSystem_t::AResultSystem_t (rank>0, %d, %d, %d)\n", numSteps, numElems, numWaves );

		blocksPerGrid = (numWaves + threadsPerBlock - 1) / threadsPerBlock;

		malloc1d(&resultsInitial,numWaves);
		malloc1d(&resultsBlocks,blocksPerGrid);

		err = hipMalloc( ( void** ) &d_wave, numWaves * sizeof( AWavePuls_t ) );
		echoError( err, "allocate the d_wave" );

		err = hipMalloc( ( void** ) &d_resultsInitial, sizeofResultSystemStep * numWaves );
		echoError( err, "allocate d_results" );

		err = hipMalloc( ( void** ) &d_resultsBlocks, sizeofResultSystemStep * blocksPerGrid );
		echoError( err, "allocate d_BlocksResult" );

		memset( resultsBlocks, 0, sizeofResultSystemStep * blocksPerGrid );//how many waves,how many initilization

		ot = OTime( t0, t1, dt );

		OSUBSYSTEMW += ot;

		//ot = OTime( t0, t1, dt );
	}
}


void AResultSystem_t::compute(
		const AWaves_t& wavesData, const ACylinderFloating_t& structData,
		const AElem_t& elemData, int myrank
) {

	oec = OElemCylinder( structData.coefAddmass, structData.coefDrag, structData.damp33,
	                     structData.diameter, elemData.d, structData.height,
	                     structData.Rho, structData.Grav
	                   );

	oec.initial_conditions( numSteps, dt, elemData.d, 0, 0, elemData.z0, 0, 0, elemData.dotz0 );

	OSUBSYSTEMW += oec;

	for ( int idW = 0; idW < numWaves; idW++ ) {

		awd = wavesData.w[idW];

		ow = OWave( awd.amp, awd.T, structData.depth, structData.vx,
		            structData.vy, structData.phi, structData.Rho, structData.Grav
		          );

		OSUBSYSTEMW += ow;

		OSUBSYSTEMW.initial_conditions( numSteps, dt, elemData.d, 0, 0, elemData.z0, 0, 0, 0 );

	}

	OSUBSYSTEMW.calc();

}


void AResultSystem_t::compute_cuda(
		const AWaves_t& wavesData, const ACylinderFloating_t& structData,
		const AElem_t& elemData, int myrank
) {


	oec = OElemCylinder( structData.coefAddmass, structData.coefDrag, structData.damp33,
	                     structData.diameter, elemData.d, structData.height,
	                     structData.Rho, structData.Grav
	                   );

	oec.initial_conditions( numSteps, dt, elemData.d, 0, 0, elemData.z0, 0, 0, elemData.dotz0 );

	OSUBSYSTEMW += oec;

	for ( int idW = 0; idW < numWaves; idW++ ) {

		awd = wavesData.w[idW];

		//printf("OWAVE ::: awd.amp=%f, awd.T=%f, structData.depth=%f, structData.vx=%f, structData.vy=%f, structData.phi=%f, structData.Rho=%f, structData.Grav=%f \n", awd.amp, awd.T, structData.depth, structData.vx, structData.vy, structData.phi, structData.Rho, structData.Grav );
		ow = OWave( awd.amp, awd.T, structData.depth, structData.vx, structData.vy, structData.phi, structData.Rho, structData.Grav );

		OSUBSYSTEMW += ow;

		OSUBSYSTEMW.initial_conditions( numSteps, dt, elemData.d, 0, 0, elemData.z0, 0, 0, 0 );

	}

	double tmp[12];
	double z, dot_z, ddot_z, F;

	//initially Fadded is 0, so dont neet to retrive for t=0, cos: ddotz=0
	for ( int idW = 0; idW < numWaves; idW++ ) {
		OSUBSYSTEMW.calc_step( idW, 0 );
		OSUBSYSTEMW.get_step_current_elem( tmp );
		//OSUBSYSTEMW.print_current_step();
		z      = tmp[2];
		dot_z  = tmp[5];
		ddot_z = tmp[8];
		F      = tmp[11];
		//printf("OSUBSYSTEMW ::: z=%f, dot_z=%f, ddot_z=%f, F=%f\n",z,dot_z,ddot_z,F);

		resultsInitial[idW].set_initial( ddot_z, dot_z, z, F );
	}
	printf( "Copy h_wave data from the host memory to the CUDA device\n" );
	err = hipMemcpy( d_wave, wavesData.w, sizeof( AWavePuls_t ) * numWaves, hipMemcpyHostToDevice );
	echoError( err, "copy from the h_wave to d_wave" );

	printf( "Copy buoy data previous step from the host memory to the CUDA device\n" );
	err = hipMemcpy( d_resultsInitial, resultsInitial, sizeof( AResultSystemStep_t ) * numWaves, hipMemcpyHostToDevice );
	echoError( err, "copy from the h_bouy to d_results" );

	ACylinderFloating_t d_structData = structData;
	AElem_t             d_elemData   = elemData;

	results[0][0].myrank = myrank;
	results[0][0].elemId = elemData.idElem;
	for ( int k = 1; k < numSteps; k++ ) {
		//printf( "kernelPerstep kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock );
		//AResultSystemStep_t* d_result, const AWavePuls_t* d_wave, const AElem_t* d_elem,const ACylinderFloating_t& cons, const int i
		kernelPerstep << < blocksPerGrid, threadsPerBlock >> > (d_resultsInitial, d_wave, d_elemData, d_structData, k);//
		err = hipGetLastError();
		echoError( err, "to launch kernelPerstep kernel" );

		//printf( "kernelReduceWave kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock );
		kernelReduceWave << < blocksPerGrid, threadsPerBlock >> > (d_resultsInitial, d_resultsBlocks);
		err = hipGetLastError();
		echoError( err, "to launch kernelReduceWave kernel" );

		err = hipMemcpy( resultsBlocks, d_resultsBlocks, sizeof( AResultSystemStep_t ) * blocksPerGrid, hipMemcpyDeviceToHost );
		echoError( err, "copy from d_BlocksResult to h_BlocksResult" );
		printf( "copy from d_BlockResult (%d) successfully!\n", blocksPerGrid );
		printf("time step = %d :::",k);
		for ( int i = 0; i < blocksPerGrid; i++ ) {
			printf_result_and_id(i,resultsBlocks[i]);
			//results[0][k] += resultsBlocks[i]; //TODO : integration over waves : sum_j^N { A_j*dom_j}
		}

	}


}

